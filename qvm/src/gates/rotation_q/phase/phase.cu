
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math.h>

extern "C" __global__ void phase_kernel(hipDoubleComplex* state, int qubit, int num_qubits, double theta) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int dim = 1 << num_qubits;
    if (i >= dim) return;

    if ((i >> qubit) & 1) {
        hipDoubleComplex phase;
        phase.x = cos(theta);  
        phase.y = sin(theta); 

        hipDoubleComplex v = state[i];
        state[i] = hipCmul(v, phase);
    }
}
