
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math.h>

extern "C" __global__ void rx_kernel(hipDoubleComplex* state, int qubit, int num_qubits, double theta) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int dim = 1 << num_qubits;
    if (i >= dim) return;

    int partner = i ^ (1 << qubit);
    if (i < partner) {
        hipDoubleComplex a = state[i];
        hipDoubleComplex b = state[partner];

        double c = cos(theta / 2.0);
        double s = sin(theta / 2.0);

        state[i].x = c * a.x - 0.0 * a.y - (0.0 * b.x + s * b.y);
        state[i].y = c * a.y + 0.0 * a.x - (s * b.x - 0.0 * b.y);

        state[partner].x = -s * a.y + c * b.x;
        state[partner].y = s * a.x + c * b.y;
    }
}
