
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math.h>

/// CUDA kernel to apply the RX(θ) gate to a single qubit.
///
/// The RX gate rotates the qubit around the X-axis on the Bloch sphere. Its matrix is:
/// ```text
/// RX(θ) = cos(θ/2) * I - i * sin(θ/2) * X
/// ```
///
/// # Parameters
/// - `state`: Quantum state vector (array of cuDoubleComplex)
/// - `qubit`: Target qubit index (0-based)
/// - `num_qubits`: Total number of qubits
/// - `theta`: Rotation angle in radians
///
/// # Note
/// - Each amplitude pair differing in the target qubit is rotated together.
extern "C" __global__ void rx_kernel(
    hipDoubleComplex* state,
    int qubit,
    int num_qubits,
    double theta
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int dim = 1 << num_qubits;
    if (i >= dim) return;

    int partner = i ^ (1 << qubit);
    if (i < partner) {
        hipDoubleComplex a = state[i];
        hipDoubleComplex b = state[partner];

        double c = cos(theta / 2.0);
        double s = sin(theta / 2.0);

        hipDoubleComplex minus_i = make_hipDoubleComplex(0.0, -1.0);
        hipDoubleComplex sin_i = hipCmul(make_hipDoubleComplex(s, 0.0), minus_i);  // -i·sin(θ/2)

        // Apply RX(θ)
        state[i]        = hipCadd(hipCmul(make_hipDoubleComplex(c, 0.0), a), hipCmul(sin_i, b));
        state[partner]  = hipCadd(hipCmul(sin_i, a), hipCmul(make_hipDoubleComplex(c, 0.0), b));
    }
}
