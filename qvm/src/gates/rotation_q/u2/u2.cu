
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math.h>

/// CUDA kernel to apply the U2(φ, λ) gate to a target qubit.
///
/// The U2 gate is defined as:
/// ```text
/// U2(φ, λ) = 1/√2 × [  1           -e^{iλ}       ]
///                       [  e^{iφ}   e^{i(φ+λ)}    ]
/// ```
///
/// # Parameters
/// - `state`: Quantum state vector (cuDoubleComplex[], size 2^n)
/// - `target`: Index of the qubit to apply the gate to
/// - `num_qubits`: Total number of qubits in the state
/// - `phi`: First phase angle (φ)
/// - `lambda`: Second phase angle (λ)
///
/// # Notes
/// - The transformation is only applied to states where the target qubit is 0.
/// - Each such state is paired with the corresponding state where the target qubit is 1.
extern "C"
__global__ void u2_kernel(
    hipDoubleComplex* state,
    int target,
    int num_qubits,
    double phi,
    double lambda
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = 1 << num_qubits;
    if (idx >= size) return;

    int mask = 1 << target;

    // Only operate on the |0⟩ component; the |1⟩ partner will be written together
    if ((idx & mask) == 0) {
        int pair = idx | mask;

        hipDoubleComplex a = state[idx];
        hipDoubleComplex b = state[pair];

        double inv_sqrt2 = 1.0 / sqrt(2.0);
        hipDoubleComplex c = make_hipDoubleComplex(inv_sqrt2, 0.0);

        hipDoubleComplex e_il  = make_hipDoubleComplex(cos(lambda), sin(lambda));
        hipDoubleComplex e_ip  = make_hipDoubleComplex(cos(phi), sin(phi));
        hipDoubleComplex e_ipl = make_hipDoubleComplex(cos(phi + lambda), sin(phi + lambda));

        hipDoubleComplex new_a = hipCsub(hipCmul(c, a), hipCmul(hipCmul(c, e_il), b));
        hipDoubleComplex new_b = hipCadd(hipCmul(hipCmul(c, e_ip), a), hipCmul(hipCmul(c, e_ipl), b));

        state[idx]  = new_a;
        state[pair] = new_b;
    }
}
