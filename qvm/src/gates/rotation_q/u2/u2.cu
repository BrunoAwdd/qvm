
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math.h>

extern "C"
__global__ void u2_kernel(hipDoubleComplex* state, int target, int num_qubits, double phi, double lambda) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = 1 << num_qubits;
    if (idx >= size) return;

    int mask = 1 << target;

    if ((idx & mask) == 0) {
        int pair = idx | mask;

        hipDoubleComplex a = state[idx];
        hipDoubleComplex b = state[pair];

        // Pré-calcula fases
        hipDoubleComplex e_il   = make_hipDoubleComplex(cos(lambda), sin(lambda));
        hipDoubleComplex e_ip   = make_hipDoubleComplex(cos(phi), sin(phi));
        hipDoubleComplex e_ipl  = make_hipDoubleComplex(cos(phi + lambda), sin(phi + lambda));
        double inv_sqrt2 = 1.0 / sqrt(2.0);
        hipDoubleComplex c = make_hipDoubleComplex(inv_sqrt2, 0.0);

        hipDoubleComplex new_a = hipCsub(hipCmul(c, a), hipCmul(hipCmul(c, e_il), b));
        hipDoubleComplex new_b = hipCadd(hipCmul(hipCmul(c, e_ip), a), hipCmul(hipCmul(c, e_ipl), b));

        state[idx] = new_a;
        state[pair] = new_b;
    }
}
