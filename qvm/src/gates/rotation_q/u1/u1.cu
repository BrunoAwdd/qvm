
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math.h>

extern "C" __global__
void u1_kernel(hipDoubleComplex* state, int target, int n, double lambda) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = 1 << n;
    if (idx >= size) return;

    int mask = 1 << target;
    if (idx & mask) {
        hipDoubleComplex e_il = make_hipDoubleComplex(cos(lambda), sin(lambda));
        state[idx] = hipCmul(e_il, state[idx]);
    }
}
