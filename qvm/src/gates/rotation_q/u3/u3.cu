
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math.h>

/// CUDA kernel to apply the U3(θ, φ, λ) gate to a single qubit.
///
/// The U3 gate is a universal single-qubit gate defined as:
/// ```text
/// U3(θ, φ, λ) =
/// [  cos(θ/2)              -e^{iλ}·sin(θ/2) ]
/// [  e^{iφ}·sin(θ/2)   e^{i(φ+λ)}·cos(θ/2) ]
/// ```
///
/// # Parameters
/// - `state`: Quantum state vector (array of cuDoubleComplex)
/// - `target`: Index of the target qubit
/// - `n`: Total number of qubits
/// - `theta`, `phi`, `lambda`: Parameters of the U3 gate in radians
///
/// # Behavior
/// - Each thread processes a pair of amplitudes (|x0⟩ and |x1⟩).
/// - Threads only process the `|0⟩` side of the pair to avoid race conditions.
extern "C" __global__
void u3_kernel(
    hipDoubleComplex* state,
    int target,
    int n,
    double theta,
    double phi,
    double lambda
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = 1 << n;

    if (idx >= size) return;

    int mask = 1 << target;

    // Only process one side of the pair
    if ((idx & mask) == 0) {
        int pair = idx | mask;

        hipDoubleComplex a = state[idx];
        hipDoubleComplex b = state[pair];

        double ct = cos(theta / 2.0);
        double st = sin(theta / 2.0);

        hipDoubleComplex e_il   = make_hipDoubleComplex(cos(lambda), sin(lambda));
        hipDoubleComplex e_ip   = make_hipDoubleComplex(cos(phi), sin(phi));
        hipDoubleComplex e_ipl  = make_hipDoubleComplex(cos(phi + lambda), sin(phi + lambda));

        hipDoubleComplex new_a = hipCsub(
            hipCmul(make_hipDoubleComplex(ct, 0.0), a),
            hipCmul(hipCmul(e_il, make_hipDoubleComplex(st, 0.0)), b)
        );

        hipDoubleComplex new_b = hipCadd(
            hipCmul(hipCmul(e_ip, make_hipDoubleComplex(st, 0.0)), a),
            hipCmul(hipCmul(e_ipl, make_hipDoubleComplex(ct, 0.0)), b)
        );

        state[idx] = new_a;
        state[pair] = new_b;
    }
}
