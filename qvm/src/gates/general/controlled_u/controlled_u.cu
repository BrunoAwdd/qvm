
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

extern "C" __global__ void controlled_u_kernel(
    hipDoubleComplex* state,
    int control,
    int target,
    int num_qubits,
    hipDoubleComplex u00,
    hipDoubleComplex u01,
    hipDoubleComplex u10,
    hipDoubleComplex u11
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int dim = 1 << num_qubits;
    if (i >= dim) return;

    // Só aplicar se controle == 1
    if (((i >> control) & 1) == 1) {
        int bit_target = (i >> target) & 1;

        int pair = i ^ (1 << target); // Flipa o bit alvo

        // Apenas metade dos índices acessa a troca para evitar duplicação
        if (i < pair) return;

        hipDoubleComplex amp_i = state[i];
        hipDoubleComplex amp_pair = state[pair];

        if (bit_target == 0) {
            state[i]     = hipCadd(hipCmul(u00, amp_i),     hipCmul(u01, amp_pair));
            state[pair]  = hipCadd(hipCmul(u10, amp_i),     hipCmul(u11, amp_pair));
        } else {
            state[i]     = hipCadd(hipCmul(u11, amp_i),     hipCmul(u10, amp_pair));
            state[pair]  = hipCadd(hipCmul(u01, amp_i),     hipCmul(u00, amp_pair));
        }
    }
}
