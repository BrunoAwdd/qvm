
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

extern "C" __global__ void s_kernel(hipDoubleComplex* state, int qubit, int num_qubits) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int dim = 1 << num_qubits;
    if (i >= dim) return;

    if ((i >> qubit) & 1) {
        hipDoubleComplex phase = make_hipDoubleComplex(0.0, 1.0); // i
        state[i] = hipCmul(state[i], phase);
    }
}
