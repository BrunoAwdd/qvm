
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math.h>

extern "C" __global__ void tdagger_kernel(hipDoubleComplex* state, int target, int num_qubits) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = 1 << num_qubits;
    if (idx >= size) return;

    if ((idx >> target) & 1) {
        double angle = -M_PI / 4.0;
        hipDoubleComplex phase = make_hipDoubleComplex(cos(angle), sin(angle));
        state[idx] = hipCmul(state[idx], phase);
    }
}
