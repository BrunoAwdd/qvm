
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

extern "C" __global__
void u3_kernel(hipDoubleComplex* state, int target, int n, double theta, double phi, double lambda) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = 1 << n;

    if (idx >= size) return;

    int mask = 1 << target;

    // Só processa pares onde o bit do qubit alvo está 0
    if ((idx & mask) == 0) {
        int pair = idx | mask;

        hipDoubleComplex a = state[idx];
        hipDoubleComplex b = state[pair];

        double ct = cos(theta / 2.0);
        double st = sin(theta / 2.0);

        hipDoubleComplex e_il = make_hipDoubleComplex(cos(lambda), sin(lambda));
        hipDoubleComplex e_ip = make_hipDoubleComplex(cos(phi), sin(phi));
        hipDoubleComplex e_ipl = make_hipDoubleComplex(cos(phi + lambda), sin(phi + lambda));

        hipDoubleComplex new_a = hipCsub(hipCmul(make_hipDoubleComplex(ct, 0.0), a),
                                       hipCmul(hipCmul(e_il, make_hipDoubleComplex(st, 0.0)), b));

        hipDoubleComplex new_b = hipCadd(hipCmul(hipCmul(e_ip, make_hipDoubleComplex(st, 0.0)), a),
                                       hipCmul(hipCmul(e_ipl, make_hipDoubleComplex(ct, 0.0)), b));

        state[idx] = new_a;
        state[pair] = new_b;
    }
}
