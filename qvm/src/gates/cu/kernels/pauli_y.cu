
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

extern "C" __global__ void pauli_y_kernel(hipDoubleComplex* state, int qubit, int num_qubits) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int dim = 1 << num_qubits;
    if (i >= dim) return;

    int partner = i ^ (1 << qubit);
    if (i < partner) {
        hipDoubleComplex a = state[i];
        hipDoubleComplex b = state[partner];

        // Apply Y gate
        state[i].x =  b.y;
        state[i].y = -b.x;

        state[partner].x = -a.y;
        state[partner].y =  a.x;
    }
}
